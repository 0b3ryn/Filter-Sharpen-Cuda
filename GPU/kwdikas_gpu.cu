
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>


hipError_t addWithCuda(int *c,  int *finalarr,  int arraySizex, int arraySizey, int size);

__global__ void addKernel(int *dev_c,  int *dev_finalarr, int arraySizex)
{
	
	int sum;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int first0;
	int first1;
	int first2;
	int first3;
	int first4;
	int first5;
	int first6;
	int first7;
	int first8;

	
		if (i < arraySizex) {
			if (j < arraySizex) {

				//ypologimos twn gwniakwn pixel
				if (i == 0 && j == 0) {
					
					first0 = 0;
					first1 = 0;
					first2 = 0;
					first3 = 0;
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = 0;
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = dev_c[(i + 1)*arraySizex + (j + 1)];
					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);
					

					if (sum < 0){
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if(sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (i == 0 && j == 511) {
					
					first0 = 0;
					first1 = 0;
					first2 = 0;
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = 0;
					first6 = dev_c[(i + 1)*arraySizex + (j - 1)];
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = 0;

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (i == 511 && j == 0) {
					
					first0 = 0;
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = dev_c[(i - 1)*arraySizex + (j + 1)];
					first3 = 0;
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = 0;
					first7 = 0;
					first8 = 0;

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (i == 511 && j == 511) {
					
					first0 = dev_c[(i - 1)*arraySizex + (j - 1)];
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = 0;
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = 0;
					first6 = 0;
					first7 = 0;
					first8 = 0;

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				} // ypologismos twn perimetrikwn pixel
				else if (i == 0 && (j != 0 || j != 511)) {
					

					first0 = 0;
					first1 = 0;
					first2 = 0;
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = dev_c[(i + 1)*arraySizex + (j - 1)];
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = dev_c[(i + 1)*arraySizex + (j - 1)];

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (i == 511 && (j != 0 || j != 511)) {
					
					first0 = dev_c[(i - 1)*arraySizex + (j - 1)];
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = dev_c[(i - 1)*arraySizex + (j + 1)];
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = 0;
					first7 = 0;
					first8 = 0;

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (j == 0 && (i != 0 || i != 511)) {
					
					first0 = 0;
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = dev_c[(i - 1)*arraySizex + (j + 1)];
					first3 = 0;
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = 0;
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = dev_c[(i + 1)*arraySizex + (j + 1)];

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else if (j == 511 && (i != 0 || i != 511)) {
					

					first0 = dev_c[(i - 1)*arraySizex + (j - 1)];
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = 0;
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = 0;
					first6 = dev_c[(i + 1)*arraySizex + (j - 1)];
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = 0;

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}

				}
				else { //ypologismos twn eswterikwn pixel

					first0 = dev_c[(i - 1)*arraySizex + (j - 1)];
					first1 = dev_c[(i - 1)*arraySizex + (j)];
					first2 = dev_c[(i - 1)*arraySizex + (j + 1)];
					first3 = dev_c[(i)*arraySizex + (j - 1)];
					first4 = dev_c[(i)*arraySizex + (j)];
					first5 = dev_c[(i)*arraySizex + (j + 1)];
					first6 = dev_c[(i + 1)*arraySizex + (j - 1)];
					first7 = dev_c[(i + 1)*arraySizex + (j)];
					first8 = dev_c[(i + 1)*arraySizex + (j + 1)];

					sum = 0;
					sum = first0 * (-1) + first1 * (-1) + first2 * (-1) + first3 * (-1) + first4 * (9) + first5 * (-1) + first6 * (-1) + first7 * (-1) + first8 * (-1);


					if (sum < 0) {
						dev_finalarr[(i)*arraySizex + (j)] = 0;
					}
					else if (sum > 255) {
						dev_finalarr[i * arraySizex + j] = 255;
					}
					else {
						dev_finalarr[i * arraySizex + j] = sum;
					}


				}




			}


		}
		


}

int main()
{
	
	FILE * pFile;

	char readChars[256];
	int arraySizex = 512;
	int arraySizey = 512;
	int size = arraySizex * arraySizey;
	int* c = (int*)malloc(sizeof(int*) * size);
	
	int* finalarr = (int*)malloc(sizeof(int*) * size);
	
	
	

	pFile = fopen("baboon.ascii.pgm", "r");
	//skip 4 lines
	fgets(readChars, 255, pFile);
	fgets(readChars, 255, pFile);
	fgets(readChars, 255, pFile);
	fgets(readChars, 255, pFile);
	//read the pixels from pgm file
	for (int i = 0; i < arraySizex; i++)
	{
		for (int j = 0; j < arraySizey; j++)
		{
			fscanf(pFile, "%s", readChars);
			int sum = atoi(readChars);
			c[i*arraySizex + j] = sum;
			
		}
	}
	fclose(pFile);
	hipError_t cudaStatus = addWithCuda(c, finalarr, arraySizex, arraySizey, size);
	free(c);
	free(finalarr);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c,  int *finalarr,  int arraySizex, int arraySizey, int size)
{	
	clock_t tic = clock();
	dim3 threadsPerBlock(32, 32,1);
	dim3 numBlocks(512 / threadsPerBlock.x, 512 / threadsPerBlock.y,1);
	
    int *dev_c;
	int *dev_finalarr;
	FILE * nFile;
    
    hipError_t cudaStatus;
	clock_t tic2 = clock();
	//metafora dedomenwn apo CPU stin GPU
	cudaStatus = hipSetDevice(0);
	cudaStatus = hipMalloc(&dev_c, size * sizeof(int));
	cudaStatus = hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMalloc(&dev_finalarr, size * sizeof(int));
	cudaStatus = hipMemcpy(dev_finalarr, finalarr, size * sizeof(int), hipMemcpyHostToDevice);
	clock_t toc2 = clock();
	printf(" %f copy data from CPU to GPU\n", (double)(toc2 - tic2) / CLOCKS_PER_SEC);
    // Launch a kernel on the GPU with one thread for each element.
	clock_t tic1 = clock();
	addKernel <<< numBlocks, threadsPerBlock >>>(dev_c, dev_finalarr, arraySizex);
	
	cudaStatus = hipGetLastError();
	//metafora dedomenwn apo GPU stin CPU
	cudaStatus = hipMemcpy(finalarr, dev_finalarr, size * sizeof(int), hipMemcpyDeviceToHost);
	clock_t toc1 = clock();
	printf(" %f Time spent in GPU\n", (double)(toc1 - tic1) / CLOCKS_PER_SEC);
	nFile = fopen("baboon1.ascii.pgm", "w");
	//write the first 4 lines in new pgm file
	fprintf(nFile, "P2\n");
	fprintf(nFile, "# baboon.pgma created by PGMA_IO::PGMA_WRITE.\n");
	fprintf(nFile, "512  512\n");
	fprintf(nFile, "255\n");
	//write the new pixels in the pgm file
	for (int i = 0; i < arraySizex; i++)
	{
		for (int j = 0; j < arraySizey; j++)
		{
			fprintf(nFile, "%d  ", finalarr[i*arraySizex + j]);
			
		}
		fprintf(nFile, "\n");
	}
	
	fclose(nFile);

	clock_t toc = clock();
	printf(" %f Total time seconds\n", (double)(toc-tic) / CLOCKS_PER_SEC);
    hipFree(dev_c);
	hipFree(dev_finalarr);
	
    
    
    return cudaStatus;
}
